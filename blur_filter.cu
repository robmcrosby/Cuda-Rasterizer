#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include "blur_filter.h"

#define BLUR_BLOCK_SIZE 16

vec3_t* hdrbuffer_get_color_at(hdrbuffer_t *buffer, int x, int y) {
   return buffer->pixels + buffer->width * y + x;
}

void blur_9X9_horizontal(hdrbuffer_t *dest, hdrbuffer_t *source) {
   int x, y, i;
   int offset[] = {0, 1, 2, 3, 4};
   float weight[] = {0.273438f, 0.21875f, 0.109375f, 0.03125f, 0.003906f};
   
   for (x = 0; x < source->width; ++x) {
      for (y = 0; y < source->height; ++y) {
         vec3_t source_color;
         vec3_t *dest_color;
         
         dest_color = hdrbuffer_get_color_at(dest, x, y);
         
         source_color = vec3_scale(hdrbuffer_get_color_at(source, x, y), weight[0]);
         *dest_color = source_color;
         
         for (i = 1; i < 5; ++i) {
            if (x+offset[i] >= source->width)
               source_color = vec3_scale(hdrbuffer_get_color_at(source, source->width-1, y), weight[i]);
            else
               source_color = vec3_scale(hdrbuffer_get_color_at(source, x+offset[i], y), weight[i]);
            *dest_color = vec3_add(dest_color, &source_color);
            
            if (x-offset[i] < 0)
               source_color = vec3_scale(hdrbuffer_get_color_at(source, 0, y), weight[i]);
            else
               source_color = vec3_scale(hdrbuffer_get_color_at(source, x-offset[i], y), weight[i]);
            *dest_color = vec3_add(dest_color, &source_color);
         }
      }
   }
}

void blur_9X9_vertical(hdrbuffer_t *dest, hdrbuffer_t *source) {
   int x, y, i;
   int offset[] = {0, 1, 2, 3, 4};
   float weight[] = {0.273438f, 0.21875f, 0.109375f, 0.03125f, 0.003906f};
   
   for (x = 0; x < source->width; ++x) {
      for (y = 0; y < source->height; ++y) {
         vec3_t source_color;
         vec3_t *dest_color;
         
         dest_color = hdrbuffer_get_color_at(dest, x, y);
         
         source_color = vec3_scale(hdrbuffer_get_color_at(source, x, y), weight[0]);
         *dest_color = source_color;
         
         for (i = 1; i < 5; ++i) {
            if (y+offset[i] >= source->height)
               source_color = vec3_scale(hdrbuffer_get_color_at(source, x, source->height-1), weight[i]);
            else
               source_color = vec3_scale(hdrbuffer_get_color_at(source, x, y+offset[i]), weight[i]);
            *dest_color = vec3_add(dest_color, &source_color);
            
            if (y-offset[i] < 0)
               source_color = vec3_scale(hdrbuffer_get_color_at(source, x, 0), weight[i]);
            else
               source_color = vec3_scale(hdrbuffer_get_color_at(source, x, y-offset[i]), weight[i]);
            *dest_color = vec3_add(dest_color, &source_color);
         }
      }
   }
}


int blur_bitmap(bitmap_t *bitmap, int itertions) {
   hdrbuffer_t buffer1, buffer2;
   color_t *color;
   vec3_t *pixel;
   int i;
   
   // create the hdr buffers.
   buffer1.pixels = (vec3_t *) malloc(bitmap->width * bitmap->height * sizeof(vec3_t));
   buffer2.pixels = (vec3_t *) malloc(bitmap->width * bitmap->height * sizeof(vec3_t));
   buffer1.width = buffer2.width = bitmap->width;
   buffer1.height = buffer2.height = bitmap->height;
   for (i = 0, color= bitmap->pixels, pixel = buffer1.pixels; i < bitmap->width * bitmap->height; ++i, ++color, ++pixel)
      *pixel = color_to_vec3(color);
   
   // apply the blur
   for (i = 0; i < itertions; ++i) {
      blur_9X9_horizontal(&buffer2, &buffer1);
      blur_9X9_vertical(&buffer1, &buffer2);
   }
   
   // put the colors back in the bitmap
   for (i = 0, color= bitmap->pixels, pixel = buffer1.pixels; i < bitmap->width * bitmap->height; ++i, ++color, ++pixel)
      *color = vec3_to_color(pixel);
   
   // free the buffers
   free(buffer1.pixels);
   free(buffer2.pixels);
   
   return 0;
}

__device__ vec3_t cuda_vec3_scale2(vec3_t *v, float s) {
   vec3_t retV;
   retV.x = v->x * s;
   retV.y = v->y * s;
   retV.z = v->z * s;
   return retV;
}

__device__ void cuda_vec3_add2(vec3_t *v1, vec3_t *v2) {
   v1->x += v2->x;
   v1->y += v2->y;
   v1->z += v2->z;
}

__device__ vec3_t* cuda_vec3_at(vec3_t *buffer, int x, int y, int width) {
   return buffer + width * y + x;
}

__constant__ int offset[] = {0, 1, 2, 3, 4};
__constant__ float weight[] = {0.273438f, 0.21875f, 0.109375f, 0.03125f, 0.003906f};

__global__ void cuda_blur_9X9_horizontal(vec3_t *buffer1, vec3_t *buffer2, int width, int height) {
   int x = blockIdx.x * blockDim.x + threadIdx.x;
   int y = blockIdx.y * blockDim.y + threadIdx.y;
   int i;
   vec3_t sum, value;
   
   if (x < width && y < height) {
      sum = cuda_vec3_scale2(cuda_vec3_at(buffer2, x, y, width), weight[0]);
      
      for (i = 1; i < 5; ++i) {
         if (x+offset[i] >= width)
            value = cuda_vec3_scale2(cuda_vec3_at(buffer2, width-1, y, width), weight[i]);
         else
            value = cuda_vec3_scale2(cuda_vec3_at(buffer2, x+offset[i], y, width), weight[i]);
         cuda_vec3_add2(&sum, &value);
         
         if (x-offset[i] < 0)
            value = cuda_vec3_scale2(cuda_vec3_at(buffer2, 0, y, width), weight[i]);
         else
            value = cuda_vec3_scale2(cuda_vec3_at(buffer2, x-offset[i], y, width), weight[i]);
         cuda_vec3_add2(&sum, &value);
      }
      
      buffer1[x + y*width] = sum;
   }
}

__global__ void cuda_blur_9X9_vertical(vec3_t *buffer1, vec3_t *buffer2, int width, int height) {
   int x = blockIdx.x * blockDim.x + threadIdx.x;
   int y = blockIdx.y * blockDim.y + threadIdx.y;
   int i;
   vec3_t sum, value;
   
   if (x < width && y < height) {
      sum = cuda_vec3_scale2(cuda_vec3_at(buffer2, x, y, width), weight[0]);
      
      for (i = 1; i < 5; ++i) {
         if (y+offset[i] >= height)
            value = cuda_vec3_scale2(cuda_vec3_at(buffer2, x, height-1, width), weight[i]);
         else
            value = cuda_vec3_scale2(cuda_vec3_at(buffer2, x, y+offset[i], width), weight[i]);
         cuda_vec3_add2(&sum, &value);
         
         if (y-offset[i] < 0)
            value = cuda_vec3_scale2(cuda_vec3_at(buffer2, x, 0, width), weight[i]);
         else
            value = cuda_vec3_scale2(cuda_vec3_at(buffer2, x, y-offset[i], width), weight[i]);
         cuda_vec3_add2(&sum, &value);
      }
      
      buffer1[x + y*width] = sum;
   }
}

void blur_9X9_horizontal_cuda(vec3_t *buffer1, vec3_t *buffer2, int width, int height) {
   dim3 block_size;
   dim3 num_blocks;
   
   block_size.x = block_size.y = BLUR_BLOCK_SIZE;
   num_blocks.x = width / block_size.x + (width % block_size.x == 0 ? 0 : 1);
   num_blocks.y = height / block_size.y + (height % block_size.y == 0 ? 0 : 1);
   
   cuda_blur_9X9_horizontal <<< num_blocks, block_size >>> (buffer1, buffer2, width, height);
}

void blur_9X9_vertical_cuda(vec3_t *buffer1, vec3_t *buffer2, int width, int height) {
   dim3 block_size;
   dim3 num_blocks;
   
   block_size.x = block_size.y = 16;
   num_blocks.x = width / block_size.x + (width % block_size.x == 0 ? 0 : 1);
   num_blocks.y = height / block_size.y + (height % block_size.y == 0 ? 0 : 1);
   
   cuda_blur_9X9_vertical <<< num_blocks, block_size >>> (buffer1, buffer2, width, height);
}

int blur_bitmap_cuda(bitmap_t *bitmap, int itertions) {
   vec3_t *buffer_h, *buffer1_d, *buffer2_d;
   color_t *color;
   vec3_t *pixel;
   size_t size;
   int i;
   
   // create the hdr buffers.
   size = bitmap->width * bitmap->height * sizeof(vec3_t);
   buffer_h = (vec3_t *) malloc(size);
   if (hipMalloc((void **) &buffer1_d, size) == hipErrorOutOfMemory)
      printf("error creating memory for blur buffer1\n");
   if (hipMalloc((void **) &buffer2_d, size) == hipErrorOutOfMemory)
      printf("error creating memory for blur buffer2\n");
   
   // convert colors to vec3_t and move to device
   for (i = 0, color= bitmap->pixels, pixel = buffer_h; i < bitmap->width * bitmap->height; ++i, ++color, ++pixel)
      *pixel = color_to_vec3(color);
   hipMemcpy(buffer1_d, buffer_h, size, hipMemcpyHostToDevice);
   
   // blur the image
   for (i = 0; i < itertions; ++i) {
      blur_9X9_horizontal_cuda(buffer2_d, buffer1_d, bitmap->width, bitmap->height);
      blur_9X9_vertical_cuda(buffer1_d, buffer2_d, bitmap->width, bitmap->height);
   }
   
   // put the colors back in the bitmap
   hipMemcpy(buffer_h, buffer1_d, size, hipMemcpyDeviceToHost);
   for (i = 0, color= bitmap->pixels, pixel = buffer_h; i < bitmap->width * bitmap->height; ++i, ++color, ++pixel)
      *color = vec3_to_color(pixel);
   
   // free the buffers.
   free(buffer_h);
   hipFree(buffer1_d);
   hipFree(buffer2_d);
   
   return 0;
}